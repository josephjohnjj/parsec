
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" 
{
    void HelloWorld_cuda_kernel(double *A_double, int k);
}

__global__ void cuda_kernel(double *A_double, int k) 
{
    int i;
    for( i = 0; i < 100; i++ )
        *(A_double+i) = *(A_double+i) + k; 
}

void HelloWorld_cuda_kernel(double *A_double, int k)
{
    cuda_kernel<<<1,1>>>(A_double, k);
}